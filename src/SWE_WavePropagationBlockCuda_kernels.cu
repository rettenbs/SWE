#include "hip/hip_runtime.h"
/**
 * @file
 * This file is part of SWE.
 *
 * @author Alexander Breuer (breuera AT in.tum.de, http://www5.in.tum.de/wiki/index.php/Dipl.-Math._Alexander_Breuer)
 *
 * @section LICENSE
 *
 * SWE is free software: you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation, either version 3 of the License, or
 * (at your option) any later version.
 *
 * SWE is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.
 *
 * You should have received a copy of the GNU General Public License
 * along with SWE.  If not, see <http://www.gnu.org/licenses/>.
 *
 *
 * @section DESCRIPTION
 *
 * CUDA Kernels for a SWE_Block, which uses solvers in the wave propagation formulation.
 */

#include "SWE_BlockCUDA.hh"
#include "SWE_WavePropagationBlockCuda_kernels.hh"

#include <cmath>
#include <cstdio>

#include "solvers/FWaveCuda.h"
#include "hipblas.h"

#define G 9.81

/**
 * The compute net-updates kernel calls the solver for a defined CUDA-Block and does a reduction over the computed wave speeds within this block.
 *
 * Remark: In overall we have nx+1 / ny+1 edges.
 *         Therefore the edges "simulation domain"/"top ghost layer" and "simulation domain"/"right ghost layer"
 *         will not be computed in a typical call of the function:
 *           computeNetUpdatesKernel<<<dimGrid,dimBlock>>>( hd, hud, hvd, bd,
 *                                                          hNetUpdatesLeftD,  hNetUpdatesRightD,
 *                                                          huNetUpdatesLeftD, huNetUpdatesRightD,
 *                                                          hNetUpdatesBelowD, hNetUpdatesAboveD,
 *                                                          hvNetUpdatesBelowD, hvNetUpdatesAboveD,
 *                                                          l_maximumWaveSpeedsD,
 *                                                          i_nX, i_nY
 *                                                        );
 *         To reduce the effect of branch-mispredictions the kernel provides optional offsets, which can be used to compute the missing edges.
 *
 *
 * @param i_h water heights (CUDA-array).
 * @param i_hu momentums in x-direction (CUDA-array).
 * @param i_hv momentums in y-direction (CUDA-array).
 * @param i_b bathymetry values (CUDA-array).
 * @param o_hNetUpdatesLeftD left going net-updates for the water height (CUDA-array).
 * @param o_hNetUpdatesRightD right going net-updates for the water height (CUDA-array).
 * @param o_huNetUpdatesLeftD left going net-updates for the momentum in x-direction (CUDA-array).
 * @param o_huNetUpdatesRightD right going net-updates for the momentum in x-direction (CUDA-array).
 * @param o_hNetUpdatesBelowD downwards going net-updates for the water height (CUDA-array).
 * @param o_hNetUpdatesAboveD upwards going net-updates for the water height (CUDA-array).
 * @param o_hvNetUpdatesBelowD downwards going net-updates for the momentum in y-direction (CUDA-array).
 * @param o_hvNetUpdatesAboveD upwards going net-updates for the momentum in y-direction (CUDA-array).
 * @param o_maximumWaveSpeeds maximum wave speed which occurred within the CUDA-block is written here (CUDA-array).
 * @param i_nX number of cells within the simulation domain in x-direction (excludes ghost layers).
 * @param i_nY number of cells within the simulation domain in y-direction (excludes ghost layers).
 * @param i_offsetX cell/edge offset in x-direction.
 * @param i_offsetY cell/edge offset in y-direction.
 */
__global__
void computeNetUpdatesKernel(
    const float* i_h, const float* i_hu, const float* i_hv, const float* i_b,
    float* o_hNetUpdatesLeftD,   float* o_hNetUpdatesRightD,
    float* o_huNetUpdatesLeftD,  float* o_huNetUpdatesRightD,
    float* o_hNetUpdatesBelowD,  float* o_hNetUpdatesAboveD,
    float* o_hvNetUpdatesBelowD, float* o_hvNetUpdatesAboveD,
    float* o_maximumWaveSpeeds,
    const int i_nX, const int i_nY,
    const int i_offsetX, const int i_offsetY,
    const int i_blockOffsetX, const int i_blockOffsetY
) {
	// Recover absolute array indices from CUDA thread constants
	// + Kernel offset
	int i = blockIdx.y * blockDim.y + threadIdx.y + i_offsetX + 1;
	int j = blockIdx.x * blockDim.x + threadIdx.x + i_offsetY + 1;

	// T is from ./solvers/FWaveCuda.h
	// this implements "typedef float T;"  by default
	T netUpdates[5];

	// computeOneDPo...(arg0,arg1,arg2) = arg0*arg2 + arg1
	// Position in h, hu, hv, b
	int unknownPosition = computeOneDPositionKernel(i,
		 j,
		 i_nY + 2);
	// Position in NetUpdate arrays (upper-right edge)
	int netUpdatePosition = computeOneDPositionKernel(i,
		j,
		i_nY + 1);
	T localMaxWaveSpeed = 0; // local maximum wave speed
	__shared__ T maxWaveSpeed[TILE_SIZE * TILE_SIZE]; // maximum wave speeds for this block

	// Returns the values of net updates in T netUpdates[5] with values
	// corresponding to ("h_left","h_right","hu_left","hu_right","MaxWaveSpeed").
	if (i_offsetY == 0) {
		int unknownPositionLeft = unknownPosition - i_nY - 2; // cell left to the current cell
		fWaveComputeNetUpdates(G,
			i_h[unknownPositionLeft],
			i_h[unknownPosition],
			i_hu[unknownPositionLeft],
			i_hu[unknownPosition],
			i_b[unknownPositionLeft],
			i_b[unknownPosition],
			netUpdates);

		int netUpdatePositionLeft = netUpdatePosition - i_nY - 1; // edge to the left
		o_hNetUpdatesLeftD[netUpdatePositionLeft] = netUpdates[0];
		o_hNetUpdatesRightD[netUpdatePositionLeft] = netUpdates[1];
		o_huNetUpdatesLeftD[netUpdatePositionLeft] = netUpdates[2];
		o_huNetUpdatesRightD[netUpdatePositionLeft] = netUpdates[3];
		localMaxWaveSpeed = netUpdates[4];
	}

	if (i_offsetX == 0) {
		int unknownPositionBelow = unknownPosition - 1;
		fWaveComputeNetUpdates(G,
			i_h[unknownPositionBelow],
			i_h[unknownPosition],
			i_hv[unknownPositionBelow],
			i_hv[unknownPosition],
			i_b[unknownPositionBelow],
			i_b[unknownPosition],
			netUpdates);

		int netUpdatePositionBelow = netUpdatePosition - 1;
		o_hNetUpdatesBelowD[netUpdatePositionBelow] = netUpdates[0];
		o_hNetUpdatesAboveD[netUpdatePositionBelow] = netUpdates[1];
		o_hvNetUpdatesBelowD[netUpdatePositionBelow] = netUpdates[2];
		o_hvNetUpdatesAboveD[netUpdatePositionBelow] = netUpdates[3];
		if (netUpdates[4] > localMaxWaveSpeed)
			localMaxWaveSpeed = netUpdates[4];
	}

	// thread1 is the id of this thread in the block
	int thread1 = threadIdx.y * blockDim.x + threadIdx.x;
	maxWaveSpeed[thread1] = localMaxWaveSpeed;

	__syncthreads();

	// Calculate the maximum wave speed of this block
	// taken from https://www.sharcnet.ca/help/index.php/CUDA_tips_and_tricks
	int nTotalThreads = blockDim.x * blockDim.y;

	while (nTotalThreads > 1)
	{
 		int halfPoint = (nTotalThreads >> 1);	// divide by two

		// only the first half of the threads will be active.
  		if (thread1 < halfPoint)
  		{
   			int thread2 = thread1 + halfPoint;

    			// Get the shared value stored by another thread

    			T temp = maxWaveSpeed[thread2];
    			if (temp > maxWaveSpeed[thread1])
       				maxWaveSpeed[thread1] = temp;

 		}

		__syncthreads();

  		// Reducing the binary tree size by two:
  		nTotalThreads = halfPoint;
	}

	o_maximumWaveSpeeds[(blockIdx.y+i_blockOffsetX) * (i_nY/TILE_SIZE+1) + (blockIdx.x+i_blockOffsetY)] = maxWaveSpeed[0];
}

/**
 * The "update unknowns"-kernel updates the unknowns in the cells with precomputed net-updates.
 *
 * @param i_hNetUpdatesLeftD left going net-updates for the water height (CUDA-array).
 * @param i_hNetUpdatesRightD right going net-updates for the water height (CUDA-array).
 * @param i_huNetUpdatesLeftD left going net-updates for the momentum in x-direction (CUDA-array).
 * @param i_huNetUpdatesRightD right going net-updates for the momentum in x-direction (CUDA-array).
 * @param i_hNetUpdatesBelowD downwards going net-updates for the water height (CUDA-array).
 * @param i_hNetUpdatesAboveD upwards going net-updates for the water height (CUDA-array).
 * @param i_hvNetUpdatesBelowD downwards going net-updates for the momentum in y-direction (CUDA-array).
 * @param i_hvNetUpdatesAboveD upwards going net-updates for the momentum in y-direction (CUDA-array).
 * @param io_h water heights (CUDA-array).
 * @param io_hu momentums in x-direction (CUDA-array).
 * @param io_hv momentums in y-direction (CUDA-array).
 * @param i_updateWidthX update width in x-direction.
 * @param i_updateWidthY update width in y-direction.
 * @param i_nX number of cells within the simulation domain in x-direction (excludes ghost layers).
 * @param i_nY number of cells within the simulation domain in y-direction (excludes ghost layers).
 */

/*
 * updateUnknownsCUBLAS
 * reexpression of updateUnknownsKernel as a sequence of hipblasSaxpy().
 * The purpose here is to take advantage of high-performance CUBLAS calls
 * in order to further disguise the cost of performing these small transactions.
 *
 * Please note that I use the index offset convention where if
 * h[i] = hnew[i+1], I write h[i-1] = hnew[i] (I find this easier to control here).
 *
 * CUDA philosophy:  think locally, act sequentially
 */
void updateUnknownsCUBLAS(
    const float* i_hNetUpdatesLeftD,   const float* i_hNetUpdatesRightD,
    const float* i_huNetUpdatesLeftD,  const float* i_huNetUpdatesRightD,
    const float* i_hNetUpdatesBelowD,  const float* i_hNetUpdatesAboveD,
    const float* i_hvNetUpdatesBelowD, const float* i_hvNetUpdatesAboveD,
    float* io_h, float* io_hu, float* io_hv,
    const float i_updateWidthX, const float i_updateWidthY,
    const int i_nX, const int i_nY )
{
	/* BIG PICTURE
	 *  h[i][j] -= dt/dx * (hNetUpdatesRight[i-1][j-1] + hNetUpdatesLeft[i][j-1])
	 *          +  dt/dy * (hNetUpdatesAbove[i-1][j-1] + hNetUpdatesBelow[i-1][j]);
	 *
	 *  hu[i][j] -= dt/dx * (huNetUpdatesRight[i-1][j-1] + huNetUpdatesLeft[i][j-1]);
	 *
         *  hv[i][j] -= dt/dy * (hvNetUpdatesAbove[i-1][j-1] + hvNetUpdatesBelow[i-1][j]);
	 *
	 *  ASSERT:  io_h, io_hv, io_hu are (i_ny + 2)x(i_nx + 2)
	 */

	//  TODO:  Use CUDA streams to obtain concurrency in the execution below.
	hipblasHandle_t cuhandle;
	hipblasCreate(&cuhandle);

	// hipblasSaxpy requires pointer to scalar, we provide them here.
	float dtdx = -i_updateWidthX;
	float dtdy = -i_updateWidthY;

	// Warning:  i here should maybe be called j, but it is already written
	// with i and I can't convince myself it should be j, so if you are
	// confused this is why.

	for(int i = 1; i <= i_nX; i++) {
		// =========================h section=========================
		// io_h[i][j] += ...
		// - (dt/dx) * hNetUpdatesRight[i-1][j-1]
		hipblasSaxpy(cuhandle, i_nY, &dtdx,
			    i_hNetUpdatesRightD + 1 + (i_nY + 1)*(i - 1), 1,
			    io_h + 1 + (i_nY + 2)*i,			  1);

		// - (dt/dx) * hNetUpdatesLeft[i][j-1]
		hipblasSaxpy(cuhandle, i_nY, &dtdx,
			    i_hNetUpdatesLeftD  + 1 + (i_nY + 1)*i, 1,
			    io_h + 1 + (i_nY + 2)*i,		    1);

		// - (dt/dy) * hNetUpdatesAbove[i-1][j-1]
		hipblasSaxpy(cuhandle, i_nY, &dtdy,
			    i_hNetUpdatesAboveD  + (i_nY + 1)*i, 1,
			    io_h + 1 + (i_nY + 2)*i,		      1);

		// - (dt/dy) * hNetUpdatesBelow[i-1][j]
		hipblasSaxpy(cuhandle, i_nY, &dtdy,
			    i_hNetUpdatesBelowD  + 1 + (i_nY + 1)*i, 1,
			    io_h + 1 + (i_nY + 2)*i,	      1);

		//=========================hu section=========================
		// - (dt/dx) * huNetUpdatesRight[i-1][j-1]
		hipblasSaxpy(cuhandle, i_nY, &dtdx,
			    i_huNetUpdatesRightD + 1 + (i_nY + 1)*(i - 1), 1,
			    io_hu + 1 + (i_nY + 2)*i,		       1);

		// - (dt/dx) * huNetUpdatesLeft[i][j-1]
		hipblasSaxpy(cuhandle, i_nY, &dtdx,
			    i_huNetUpdatesLeftD  + 1 + (i_nY + 1)*i, 1,
			    io_hu + 1 + (i_nY + 2)*i,		       1);


		//=========================hv section=========================
		// - (dt/dy) * hvNetUpdatesAbove[i-1][j-1]
		hipblasSaxpy(cuhandle, i_nY, &dtdy,
			    i_hvNetUpdatesAboveD  + (i_nY + 1)*i, 1,
			    io_hv + 1 + (i_nY + 2)*i,		       1);

		// - (dt/dy) * hvNetUpdatesBelow[i-1][j]);
		hipblasSaxpy(cuhandle, i_nY, &dtdy,
			    i_hvNetUpdatesBelowD  + 1 + (i_nY + 1)*i, 1,
			    io_hv + 1 + (i_nY + 2)*i,	       1);
	}


	hipDeviceSynchronize();
	hipblasDestroy(cuhandle);


}

__global__
void updateUnknownsKernel(
    const float* i_hNetUpdatesLeftD,   const float* i_hNetUpdatesRightD,
    const float* i_huNetUpdatesLeftD,  const float* i_huNetUpdatesRightD,
    const float* i_hNetUpdatesBelowD,  const float* i_hNetUpdatesAboveD,
    const float* i_hvNetUpdatesBelowD, const float* i_hvNetUpdatesAboveD,
    float* io_h, float* io_hu, float* io_hv,
    const float i_updateWidthX, const float i_updateWidthY,
    const int i_nX, const int i_nY )
{
	int i = blockIdx.x * TILE_SIZE + threadIdx.x;
	int j = blockIdx.y * TILE_SIZE + threadIdx.y;

	// Position in h, hu, hv, b
	int oneDPosition = computeOneDPositionKernel(i+1, j+1, i_nY + 2);
	// Position in *NetUpdates*
	int netUpdatePosition = computeOneDPositionKernel(i+1, j+1, i_nY + 1);

	// h updates as the sum of x- and y- positions
	io_h[oneDPosition] -=
                          i_updateWidthX * (i_hNetUpdatesRightD[netUpdatePosition - i_nY - 1] + i_hNetUpdatesLeftD[netUpdatePosition])
			+ i_updateWidthY * (i_hNetUpdatesAboveD[netUpdatePosition - 1] + i_hNetUpdatesBelowD[netUpdatePosition]);

	// hu contains only x component data, so it updates from the left and right
	io_hu[oneDPosition] -= i_updateWidthX * (i_huNetUpdatesRightD[netUpdatePosition - i_nY - 1] + i_huNetUpdatesLeftD[netUpdatePosition]);

	// hv contains ony y component data, so it updates from the top and bottom
	io_hv[oneDPosition] -= i_updateWidthY * (i_hvNetUpdatesAboveD[netUpdatePosition - 1] + i_hvNetUpdatesBelowD[netUpdatePosition]);
}

/**
 * Compute the position of 2D coordinates in a 1D array.
 *   array[i][j] -> i * ny + j
 *
 * @param i_i row index.
 * @param i_j column index.
 * @param i_nY #(cells in y-direction).
 * @return 1D index.
 */
__device__
inline int computeOneDPositionKernel(const int i_i, const int i_j, const int i_nY) {
  return i_i*i_nY + i_j;
}
