#include "hip/hip_runtime.h"
/**
 * @file
 * This file is part of SWE.
 *
 * @author Alexander Breuer (breuera AT in.tum.de, http://www5.in.tum.de/wiki/index.php/Dipl.-Math._Alexander_Breuer)
 *
 * @section LICENSE
 *
 * SWE is free software: you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation, either version 3 of the License, or
 * (at your option) any later version.
 *
 * SWE is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.
 *
 * You should have received a copy of the GNU General Public License
 * along with SWE.  If not, see <http://www.gnu.org/licenses/>.
 *
 *
 * @section DESCRIPTION
 *
 * CUDA Kernels for a SWE_Block, which uses solvers in the wave propagation formulation.
 */

#include "SWE_BlockCUDA.hh"
#include "SWE_WavePropagationBlockCuda_kernels.hh"

#include <cmath>
#include <cstdio>

#include "solvers/FWaveCuda.h"

#define MAX(a,b) (a > b ? a : b)
#define G 9.81

/**
 * The compute net-updates kernel calls the solver for a defined CUDA-Block and does a reduction over the computed wave speeds within this block.
 *
 * Remark: In overall we have nx+1 / ny+1 edges.
 *         Therefore the edges "simulation domain"/"top ghost layer" and "simulation domain"/"right ghost layer"
 *         will not be computed in a typical call of the function:
 *           computeNetUpdatesKernel<<<dimGrid,dimBlock>>>( hd, hud, hvd, bd,
 *                                                          hNetUpdatesLeftD,  hNetUpdatesRightD,
 *                                                          huNetUpdatesLeftD, huNetUpdatesRightD,
 *                                                          hNetUpdatesBelowD, hNetUpdatesAboveD,
 *                                                          hvNetUpdatesBelowD, hvNetUpdatesAboveD,
 *                                                          l_maximumWaveSpeedsD,
 *                                                          i_nx, i_ny
 *                                                        );
 *         To reduce the effect of branch-mispredictions the kernel provides optional offsets, which can be used to compute the missing edges.
 *
 *
 * @param i_h water heights (CUDA-array).
 * @param i_hu momentums in x-direction (CUDA-array).
 * @param i_hv momentums in y-direction (CUDA-array).
 * @param i_b bathymetry values (CUDA-array).
 * @param o_hNetUpdatesLeftD left going net-updates for the water height (CUDA-array).
 * @param o_hNetUpdatesRightD right going net-updates for the water height (CUDA-array).
 * @param o_huNetUpdatesLeftD left going net-updates for the momentum in x-direction (CUDA-array).
 * @param o_huNetUpdatesRightD right going net-updates for the momentum in x-direction (CUDA-array).
 * @param o_hNetUpdatesBelowD downwards going net-updates for the water height (CUDA-array).
 * @param o_hNetUpdatesAboveD upwards going net-updates for the water height (CUDA-array).
 * @param o_hvNetUpdatesBelowD downwards going net-updates for the momentum in y-direction (CUDA-array).
 * @param o_hvNetUpdatesAboveD upwards going net-updates for the momentum in y-direction (CUDA-array).
 * @param o_maximumWaveSpeeds maximum wave speed which occurred within the CUDA-block is written here (CUDA-array).
 * @param i_nx number of cells within the simulation domain in x-direction (excludes ghost layers).
 * @param i_ny number of cells within the simulation domain in y-direction (excludes ghost layers).
 * @param i_offsetX cell/edge offset in x-direction.
 * @param i_offsetY cell/edge offset in y-direction.
 */
__global__
void computeNetUpdatesKernel(
    const float* i_h, const float* i_hu, const float* i_hv, const float* i_b,
    float* o_hNetUpdatesLeftD,   float* o_hNetUpdatesRightD,
    float* o_huNetUpdatesLeftD,  float* o_huNetUpdatesRightD,
    float* o_hNetUpdatesBelowD,  float* o_hNetUpdatesAboveD,
    float* o_hvNetUpdatesBelowD, float* o_hvNetUpdatesAboveD,
    float* o_maximumWaveSpeeds,
    const int i_nX, const int i_nY,
    const int i_offsetX, const int i_offsetY,
    const int i_blockOffsetX, const int i_blockOffsetY
) {
	// Recover absolute array indices from CUDA thread constants
	int i = blockIdx.x * TILE_SIZE + threadIdx.x;
	int j = blockIdx.y * TILE_SIZE + threadIdx.y;

	// T is from ./solvers/FWaveCuda.h
	// this implements "typedef float T;"  by default
	T netUpdates[5];

	
	// computeOneDPo...(arg0,arg1,arg2) = arg0*arg2 + arg1
	int oneDPosition = computeOneDPositionKernel(i + i_offsetX + i_blockOffsetX,
		 j + i_offsetY + i_blockOffsetY,
		 i_nY + 2);
	int maxWaveSpeedPos = computeOneDPositionKernel(i, j, i_nY+1);

	// Returns the values of net updates in T netUpdates[5] with values
	// corresponding to ("h_left","h_right","hu_left","hu_right","MaxWaveSpeed").
	fWaveComputeNetUpdates(G,
		i_h[oneDPosition - i_nY - 2],
		i_h[oneDPosition],
		i_hu[oneDPosition - i_nY - 2],
		i_hu[oneDPosition],
		i_b[oneDPosition - i_nY - 2],
		i_b[oneDPosition],
		netUpdates);

	o_hNetUpdatesLeftD[oneDPosition - i_nY - 2] = netUpdates[0];
	o_hNetUpdatesRightD[oneDPosition] = netUpdates[1];
	o_huNetUpdatesLeftD[oneDPosition - i_nY - 2] = netUpdates[2];
	o_huNetUpdatesRightD[oneDPosition] = netUpdates[3];
	o_maximumWaveSpeeds[maxWaveSpeedPos] = netUpdates[4];

	fWaveComputeNetUpdates(G,
		i_h[oneDPosition - 1],
		i_h[oneDPosition],
		i_hv[oneDPosition - 1],
		i_hv[oneDPosition],
		i_b[oneDPosition - 1],
		i_b[oneDPosition],
		netUpdates);

	o_hNetUpdatesBelowD[oneDPosition - 1] = netUpdates[0];
	o_hNetUpdatesAboveD[oneDPosition] = netUpdates[1];
	o_hvNetUpdatesBelowD[oneDPosition - 1] = netUpdates[2];
	o_hvNetUpdatesAboveD[oneDPosition] = netUpdates[3];
	o_maximumWaveSpeeds[maxWaveSpeedPos] = MAX(o_maximumWaveSpeeds[maxWaveSpeedPos], netUpdates[4]);
}

/**
 * The "update unknowns"-kernel updates the unknowns in the cells with precomputed net-updates.
 *
 * @param i_hNetUpdatesLeftD left going net-updates for the water height (CUDA-array).
 * @param i_hNetUpdatesRightD right going net-updates for the water height (CUDA-array).
 * @param i_huNetUpdatesLeftD left going net-updates for the momentum in x-direction (CUDA-array).
 * @param i_huNetUpdatesRightD right going net-updates for the momentum in x-direction (CUDA-array).
 * @param i_hNetUpdatesBelowD downwards going net-updates for the water height (CUDA-array).
 * @param i_hNetUpdatesAboveD upwards going net-updates for the water height (CUDA-array).
 * @param i_hvNetUpdatesBelowD downwards going net-updates for the momentum in y-direction (CUDA-array).
 * @param i_hvNetUpdatesAboveD upwards going net-updates for the momentum in y-direction (CUDA-array).
 * @param io_h water heights (CUDA-array).
 * @param io_hu momentums in x-direction (CUDA-array).
 * @param io_hv momentums in y-direction (CUDA-array).
 * @param i_updateWidthX update width in x-direction.
 * @param i_updateWidthY update width in y-direction.
 * @param i_nx number of cells within the simulation domain in x-direction (excludes ghost layers).
 * @param i_ny number of cells within the simulation domain in y-direction (excludes ghost layers).
 */
__global__
void updateUnknownsKernel(
    const float* i_hNetUpdatesLeftD,   const float* i_hNetUpdatesRightD,
    const float* i_huNetUpdatesLeftD,  const float* i_huNetUpdatesRightD,
    const float* i_hNetUpdatesBelowD,  const float* i_hNetUpdatesAboveD,
    const float* i_hvNetUpdatesBelowD, const float* i_hvNetUpdatesAboveD,
    float* io_h, float* io_hu, float* io_hv,
    const float i_updateWidthX, const float i_updateWidthY,
    const int i_nX, const int i_nY )
{
	int i = blockIdx.x * TILE_SIZE + threadIdx.x;
	int j = blockIdx.y * TILE_SIZE + threadIdx.y;

	// TODO I think we also need the blockOffset here ...
	int oneDPosition = computeOneDPositionKernel(i, j, i_nY + 2);

	io_h[oneDPosition] -= i_updateWidthX * (i_hNetUpdatesRightD[oneDPosition - i_nY - 2]
			+ i_hNetUpdatesLeftD[oneDPosition])
		+ i_updateWidthY * (i_hNetUpdatesAboveD[oneDPosition - i_nY - 2]
			+ i_hNetUpdatesBelowD[oneDPosition]);
	io_hu[oneDPosition] -= i_updateWidthX * (i_huNetUpdatesRightD[oneDPosition - i_nY - 2]
		+ i_huNetUpdatesLeftD[oneDPosition]);
	io_hv[oneDPosition] -= i_updateWidthY * (i_hvNetUpdatesAboveD[oneDPosition - i_nY - 2]
		+ i_hvNetUpdatesBelowD[oneDPosition]);
}

/**
 * Compute the position of 2D coordinates in a 1D array.
 *   array[i][j] -> i * ny + j
 *
 * @param i_i row index.
 * @param i_j column index.
 * @param i_ny #(cells in y-direction).
 * @return 1D index.
 */
__device__
inline int computeOneDPositionKernel(const int i_i, const int i_j, const int i_ny) {
  return i_i*i_ny + i_j;
}
